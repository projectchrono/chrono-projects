
#include <hip/hip_runtime.h>
#if defined(__CUDA_ARCH__)
    #include "chrono/multicore_math/other_types.h"
    #include "chrono/multicore_math/simd_non.h"

namespace chrono {

CUDA_HOST_DEVICE vec3 operator-(const vec3& a, const vec3& b) {
    return simd::Sub(a, b);
}
CUDA_HOST_DEVICE vec3 operator-(const vec3& a, const int& b) {
    return simd::Sub(a, simd::Set(b));
}
CUDA_HOST_DEVICE vec3 operator+(const vec3& a, const vec3& b) {
    return simd::Add(a, b);
}
CUDA_HOST_DEVICE vec3 operator+(const vec3& a, const int& b) {
    return simd::Add(a, simd::Set(b));
}
CUDA_HOST_DEVICE vec3 Clamp(const vec3& a, const vec3& clamp_min, const vec3& clamp_max) {
    return simd::Max(clamp_min, simd::Min(a, clamp_max));
}
}  // namespace chrono
#endif
